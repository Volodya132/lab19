﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>

using namespace std;

#define INF 999999
#define MAX_NODES 100
#define WIDTH 800
#define HEIGHT 800
#define MAX_ITERATIONS 10000

//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__device__ unsigned char computePixel(float x, float y, float a) {
	float suma = 0;
	float lastX = x;

	for (int i = 0; i < MAX_ITERATIONS; i++)
	{
		float newX = a * lastX * (1 - lastX);
		suma += logf(fabsf(a * (1 - 2 * lastX)));

		if (i > 100)
		{
			if (fabsf(newX - lastX) < 1e-6)
			{
				return (unsigned char)(suma * 255.0 / MAX_ITERATIONS);
			}
		}

		lastX = newX;
	}

	return 0;
}

__global__ void fractal(unsigned char* image, float aMin, float aMax, float bMin, float bMax, float dx, float dy) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float a = aMin + col * dx;
	float b = bMin + row * dy;
	float x = 0.5;
	float y = 0.5;

	unsigned char value = computePixel(x, y, a);
	image[row * WIDTH + col] = value;
}

int main() {
	printf("Krysa Volodymyr: \n");
	printf("19 laba - ");
	unsigned char *image;
	hipMallocManaged(&image, WIDTH * HEIGHT * sizeof(unsigned char));
	float aMin = 2.4, aMax = 4.0, bMin = 0.1, bMax = 0.9;
	float dx = (aMax - aMin) / WIDTH;
	float dy = (bMax - bMin) / HEIGHT;

	dim3 blocks(WIDTH / 16, HEIGHT / 16);
	dim3 threads(16,16);

	fractal<<<blocks, threads>>>(image, aMin, aMax, bMin, bMax, dx, dy);
	hipDeviceSynchronize();

	FILE *file = fopen("fractal.png", "wb");
	fprintf(file, "P5\n%d %d\n255\n", WIDTH, HEIGHT);
	//fwrite(image,sizeof(unsigned char), WIDTH * HEIGHT, file);
	fclose(file);
	hipFree(image);
	return 0;
}